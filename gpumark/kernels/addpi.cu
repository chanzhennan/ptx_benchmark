#include "../op.h"

using namespace std;

#define CUDA_CHECK(ans) \
  { cuda_assert((ans), __FILE__, __LINE__); }

inline void cuda_assert(hipError_t code, const char* file, int line) {
  if (code != hipSuccess) {
    const char* err_str = nullptr;
    hipDrvGetErrorString(code, &err_str);
    fprintf(stderr, "CUDA Error: %s %s %d\n", err_str, file, line);
    exit(code);
  }
}

void AddPi() {
  int result[4];

  hipDevice_t device;
  hipCtx_t context;
  hipModule_t module;
  hipFunction_t function;
  char moduleFile[] =
      "/share/chenzhennan/sourceCode/cuDa/ptx_benchmark/gpumark/kernels/"
      "addpi.ptx";
  char kernelName[] = "AddPi";

  hipError_t r = hipSuccess;

  hipError_t err = hipInit(0);
  hipDeviceGet(&device, 0);
  hipCtxCreate(&context, 0, device);

  CUDA_CHECK(hipModuleLoad(&module, moduleFile));
  CUDA_CHECK(hipModuleGetFunction(&function, module, kernelName));

  int size = 4;
  unsigned int byteSize = size * sizeof(int);
  int* h_a = (int*)malloc(byteSize);
  hipDeviceptr_t d_a;
  hipMalloc(&d_a, byteSize);

  for (int i = 0; i < size; i++) h_a[i] = i;

  hipMemcpyHtoD(d_a, h_a, byteSize);

  int64_t a = 10;

  void* args[]{&a};
  // hipModuleLaunchKernel ( hipFunction_t f, unsigned int  gridDimX, unsigned int
  // gridDimY, unsigned int  gridDimZ, unsigned int  blockDimX, unsigned int
  // blockDimY, unsigned int  blockDimZ, unsigned int  sharedMemBytes, hipStream_t
  // hStream, void** kernelParams, void** extra )
  hipModuleLaunchKernel(function, 1, 1, 1, size, 1, 1, 0, 0, (void**)args, 0);

  hipCtxSynchronize();
}
